/*
 * 矩阵乘法
 * 矩阵A(m, k) * 矩阵B(k, n) = 矩阵C(m, n)
 * 矩阵C的计算方式：C[i][j] = sum(A[i][k] * B[k][j]) for k in range(k)
 * 矩阵C的计算方式：C[i][j] = sum(A[i][k] * B[k][j]) for k in range(k)
 * 矩阵C的计算方式：C[i][j] = sum(A[i][k] * B[k][j]) for k in range(k)
 * 矩阵C的计算方式：C[i][j] = sum(A[i][k] * B[k][j]) for k in range(k)
 */

#include <iostream>
#include <hip/hip_runtime.h>

#include <cstdlib>

using namespace std;

void random_matrix(int rows, int cols, float *h_A) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            h_A[(i) * (cols) + (j)]  = rand() / (float)RAND_MAX;
        }
    }
} 

void cpu_sgemm(float *A, float *B, float *C, const int M, const int N, const int K) {
    for (int n = 0; n < N; n++) {
        for (int m = 0; m < M; m++) {
            for (int k = 0; k < K; k++) {
                C[m * N + n] += A[m * K + k] * B[k * N + n];
            }
        }
    }
} 

float compare_matrix(float *A, float *B, const int M, const int N) {
    int i, j;
    float max_diff = 0.0f, diff;
    int printed = 0;
    for (i = 0; i < M; i++) {
        for (j = 0; j < N; j++) {
            diff = abs(A[i * N + j] - B[i * N + j]);
            max_diff = max(max_diff, diff);
            if (printed == 0) {
                if (max_diff > 0.5f || max_diff < -0.5f){
                    cout << "A[" << i << "][" << j << "] = " << A[i * N + j] << ", B[" << i << "][" << j << "] = " << B[i * N + j] << ", diff = " << diff << endl;
                    printed++;
                }
            }
        }
    }
    return max_diff;
}

#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4 *>(&(pointer))[0])

/*
C[y][x]= 
k=0
∑
K−1
​
 A[y][k]⋅B[k][x]

 A: M * K
 B: K * N
 C: M * N
*/
template<unsigned int M_NUM_PER_BLOCK, unsigned int N_NUM_PER_BLOCK, unsigned int K_NUM_PER_BLOCK, unsigned int NUM_PER_THREAD>
__global__ void sgemm_kernel(float *A, float *B, float *C, const int M, const int N, const int K) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    float* A_row_start = A + blockIdx.y * M_NUM_PER_BLOCK * K;
    float* B_col_start = B + blockIdx.x * N_NUM_PER_BLOCK;

    __shared__ float shared_A[M_NUM_PER_BLOCK][K_NUM_PER_BLOCK];
    __shared__ float shared_B[K_NUM_PER_BLOCK][N_NUM_PER_BLOCK];

    float sum[NUM_PER_THREAD]= {0.0f};
    for (int k = 0; k < K; k += K_NUM_PER_BLOCK) {
        FETCH_FLOAT4(shared_A[ty][tx * NUM_PER_THREAD]) = FETCH_FLOAT4(A_row_start[k + ty * K + tx * NUM_PER_THREAD]);
        FETCH_FLOAT4(shared_B[ty][tx * NUM_PER_THREAD]) = FETCH_FLOAT4(B_col_start[N *(ty+k) + tx * NUM_PER_THREAD]);
        __syncthreads(); 
        for (int n = 0; n < NUM_PER_THREAD; n++) {
            for (int k = 0; k < K_NUM_PER_BLOCK; k++) {
                sum[n] += shared_A[ty][k] * shared_B[k][tx * NUM_PER_THREAD + n];
            }
        }
        __syncthreads();
    }

    float *C_ptr = C + blockIdx.y * N * M_NUM_PER_BLOCK + blockIdx.x * N_NUM_PER_BLOCK;
    for (int n = 0; n < NUM_PER_THREAD; n++) {
        C_ptr[ty * N + tx * NUM_PER_THREAD + n] = sum[n];
    }
}   

int main() {
    const int m = 512;
    const int n = 512;
    const int k = 512;
    const size_t mem_size_A = m * k * sizeof(float);
    const size_t mem_size_B = k * n * sizeof(float);
    const size_t mem_size_C = m * n * sizeof(float);

    float *h_A = (float *)malloc(mem_size_A);
    float *h_B = (float *)malloc(mem_size_B);

    float *matrix_C_host_gpu_calc = (float *)malloc(mem_size_C);
    float *matrix_C_host_cpu_calc = (float *)malloc(mem_size_C);

    random_matrix(m, k, h_A);
    random_matrix(k, n, h_B);
    memset(matrix_C_host_gpu_calc, 0, mem_size_C);
    memset(matrix_C_host_cpu_calc, 0, mem_size_C); 

    float *d_A;
    float *d_B;
    float *d_C;

    hipMalloc((void**)&d_A, mem_size_A);
    hipMalloc((void**)&d_B, mem_size_B);
    hipMalloc((void**)&d_C, mem_size_C);


    hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

    cpu_sgemm(h_A, h_B, matrix_C_host_cpu_calc, m, n, k);

    dim3 block(8, 32);
    dim3 grid((m / 32), (n / 32));

    constexpr unsigned int M_NUM_PER_BLOCK = 32;
    constexpr unsigned int N_NUM_PER_BLOCK = 32;
    constexpr unsigned int K_NUM_PER_BLOCK = 32;
    constexpr unsigned int NUM_PER_THREAD = 4;
    

    sgemm_kernel<M_NUM_PER_BLOCK, N_NUM_PER_BLOCK, K_NUM_PER_BLOCK, NUM_PER_THREAD><<<grid, block>>>(d_A, d_B, d_C, m, n, k);
    hipMemcpy(matrix_C_host_gpu_calc, d_C, mem_size_C, hipMemcpyDeviceToHost);

    float diff = compare_matrix(matrix_C_host_gpu_calc, matrix_C_host_cpu_calc, m, n);
    if (diff > 0.5f || diff < -0.5f) {
        cout << "diff: " << diff << endl;
    } else {
        cout << "success" << endl;
    }
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    return 0;
}