#include <iostream>
#include <hip/hip_runtime.h>



using namespace std;

// Error checking macro
void cudaCheckError() { 
    hipError_t err = hipGetLastError(); 
    if(err != hipSuccess) { 
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl; 
        exit(-1); 
    } 
}

const int THREADS_PER_BLOCK = 256;

__global__ void reduce(float *input, float *output) {

    int tid = threadIdx.x;
    volatile __shared__ float shared_input[THREADS_PER_BLOCK];  //volatile 是什么意思
    float *input_begin = input + blockIdx.x * blockDim.x * 2; 
    shared_input[tid] = input_begin[tid] + input_begin[tid + blockDim.x];
    __syncthreads();

    for (int i = blockDim.x / 2 ; i > 32; i /= 2) {
        if (tid < i) { 
            shared_input[tid] += shared_input[tid + i];
        }
        __syncthreads();
    }

    if (tid < 32) {
        shared_input[tid] += shared_input[tid + 32];
        shared_input[tid] += shared_input[tid + 16];
        shared_input[tid] += shared_input[tid + 8];
        shared_input[tid] += shared_input[tid + 4];
        shared_input[tid] += shared_input[tid + 2];
        shared_input[tid] += shared_input[tid + 1];
    } 

    if (tid == 0) {
        output[blockIdx.x] = shared_input[0];
    }
} 


int main() {
    const int N = 32 * 1024 * 1024;
    float *input = (float*)malloc(N * sizeof(float));
    float *d_input;
    hipMalloc((void**)&d_input, N * sizeof(float));

    int block_num = N / THREADS_PER_BLOCK / 2;
    float *output = (float*)malloc(block_num * sizeof(float));
    float *d_output;
    hipMalloc((void**)&d_output, 2 * block_num * sizeof(float));

    float *result = (float*)malloc(2 * block_num * sizeof(float));

    for (int i = 0; i < N; i++) {
        input[i] = 2.0 * (float)rand() / (float)RAND_MAX - 1.0;
    }

    // cpu reduce
    for (int i = 0; i < block_num; i++) {
        float sum = 0;
        for (int j = 0; j < 2 * THREADS_PER_BLOCK; j++) {
            sum += input[i * THREADS_PER_BLOCK * 2 + j];
        }
        result[i] = sum;
    }

    cout << "cpu reduce done" << endl;
    cout << "result[0] = " << result[0] << endl;
    cout << "result[1] = " << result[1] << endl;
    cout << "result[2] = " << result[2] << endl;
    cout << "result[3] = " << result[3] << endl;

    // gpu reduce
    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 block(THREADS_PER_BLOCK);
    dim3 grid(block_num);

    reduce<<<grid, block>>>(d_input, d_output); // 所以grid 表示最上层结构， 一个gpu表示一个grid?
    hipMemcpy(output, d_output, block_num * sizeof(float), hipMemcpyDeviceToHost);
    cout << "gpu reduce done" << endl;
    cout << "output[0] = " << output[0] << endl;
    cout << "output[1] = " << output[1] << endl;
    cout << "output[2] = " << output[2] << endl;
    cout << "output[3] = " << output[3] << endl;

    hipFree(d_input);
    hipFree(d_output);
    free(input);
    free(output);
    free(result);
    return 0;

}
