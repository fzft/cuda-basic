/*
 * 矩阵乘法
 * 矩阵A(m, k) * 矩阵B(k, n) = 矩阵C(m, n)
 * 矩阵C的计算方式：C[i][j] = sum(A[i][k] * B[k][j]) for k in range(k)
 * 矩阵C的计算方式：C[i][j] = sum(A[i][k] * B[k][j]) for k in range(k)
 * 矩阵C的计算方式：C[i][j] = sum(A[i][k] * B[k][j]) for k in range(k)
 * 矩阵C的计算方式：C[i][j] = sum(A[i][k] * B[k][j]) for k in range(k)
 */

#include <iostream>
#include <hip/hip_runtime.h>

#include <cstdlib>

using namespace std;

void random_matrix(int rows, int cols, float *h_A) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            h_A[(i) * (cols) + (j)]  = rand() / (float)RAND_MAX;
        }
    }
} 

void cpu_sgemm(float *A, float *B, float *C, const int M, const int N, const int K) {
    for (int n = 0; n < N; n++) {
        for (int m = 0; m < M; m++) {
            for (int k = 0; k < K; k++) {
                C[m * N + n] += A[m * K + k] * B[k * N + n];
            }
        }
    }
} 

float compare_matrix(float *A, float *B, const int M, const int N) {
    int i, j;
    float max_diff = 0.0f, diff;
    int printed = 0;
    for (i = 0; i < M; i++) {
        for (j = 0; j < N; j++) {
            diff = abs(A[i * N + j] - B[i * N + j]);
            max_diff = max(max_diff, diff);
            if (printed == 0) {
                if (max_diff > 0.5f || max_diff < -0.5f){
                    cout << "A[" << i << "][" << j << "] = " << A[i * N + j] << ", B[" << i << "][" << j << "] = " << B[i * N + j] << ", diff = " << diff << endl;
                    printed++;
                }
            }
        }
    }
    return max_diff;
}

/*
C[y][x]= 
k=0
∑
K−1
​
 A[y][k]⋅B[k][x]
*/
template<unsigned int BLOCK_SIZE, unsigned int STRIDE>
__global__ void sgemm_kernel(float *A, float *B, float *C, const int M, const int N, const int K) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    float *A_row_start = A + blockIdx.y * blockDim.y * K * STRIDE; 
    float *B_col_start = B + blockIdx.x * blockDim.x * STRIDE;

    __shared__ float shared_A[BLOCK_SIZE * STRIDE][BLOCK_SIZE * STRIDE];
    __shared__ float shared_B[BLOCK_SIZE * STRIDE][BLOCK_SIZE * STRIDE];

    float sum[STRIDE][STRIDE] = {0.0f};
    // tile by tile
    for (int s = 0; s < K; s+= BLOCK_SIZE * STRIDE) {
        for (int i = 0; i < STRIDE; i++) {
            for (int j = 0; j < STRIDE; j++) {
                shared_A[threadIdx.y + i * BLOCK_SIZE][threadIdx.x + j * BLOCK_SIZE] = A_row_start[(threadIdx.y + i * BLOCK_SIZE) * K + s + j * BLOCK_SIZE + threadIdx.x];
                shared_B[threadIdx.y + i * BLOCK_SIZE][threadIdx.x + j * BLOCK_SIZE] = B_col_start[(threadIdx.y + i * BLOCK_SIZE + s) * N  + j * BLOCK_SIZE + threadIdx.x];
            }
        }
        __syncthreads();
       for (int i = 0; i < STRIDE; i++) {
            for (int j = 0; j < STRIDE; j++) {
                for (int k = 0; k < BLOCK_SIZE * STRIDE; k++) {
                    sum[i][j] += shared_A[threadIdx.y + i * BLOCK_SIZE][k] * shared_B[k][threadIdx.x + j * BLOCK_SIZE];
                }
            }
        }
        __syncthreads();
    } //表示同时计算 C中 STRIDE × STRIDE 个位置的值

    float *C_row_start = C + blockIdx.y * blockDim.y * N * STRIDE + blockIdx.x * blockDim.x * STRIDE;
    for (int i = 0; i < STRIDE; i++) {
        for (int j = 0; j < STRIDE; j++) {
            C_row_start[(threadIdx.y + i * BLOCK_SIZE) * N + (threadIdx.x + j * BLOCK_SIZE)] = sum[i][j];
        }
    }
}  

int main() {
    const int m = 512;
    const int n = 512;
    const int k = 512;
    const size_t mem_size_A = m * k * sizeof(float);
    const size_t mem_size_B = k * n * sizeof(float);
    const size_t mem_size_C = m * n * sizeof(float);

    float *h_A = (float *)malloc(mem_size_A);
    float *h_B = (float *)malloc(mem_size_B);

    float *matrix_C_host_gpu_calc = (float *)malloc(mem_size_C);
    float *matrix_C_host_cpu_calc = (float *)malloc(mem_size_C);

    random_matrix(m, k, h_A);
    random_matrix(k, n, h_B);
    memset(matrix_C_host_gpu_calc, 0, mem_size_C);
    memset(matrix_C_host_cpu_calc, 0, mem_size_C); 

    float *d_A;
    float *d_B;
    float *d_C;

    hipMalloc((void**)&d_A, mem_size_A);
    hipMalloc((void**)&d_B, mem_size_B);
    hipMalloc((void**)&d_C, mem_size_C);


    hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

    cpu_sgemm(h_A, h_B, matrix_C_host_cpu_calc, m, n, k);

    const int threads_per_block = 16;
    constexpr int STRIDE = 2;
    dim3 block(threads_per_block, threads_per_block);
    dim3 grid((m + threads_per_block - 1) / threads_per_block / STRIDE, (n + threads_per_block - 1) / threads_per_block / STRIDE);
    

    sgemm_kernel<threads_per_block, STRIDE><<<grid, block>>>(d_A, d_B, d_C, m, n, k);
    hipMemcpy(matrix_C_host_gpu_calc, d_C, mem_size_C, hipMemcpyDeviceToHost);

    float diff = compare_matrix(matrix_C_host_gpu_calc, matrix_C_host_cpu_calc, m, n);
    if (diff > 0.5f || diff < -0.5f) {
        cout << "diff: " << diff << endl;
    } else {
        cout << "success" << endl;
    }
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    return 0;
}