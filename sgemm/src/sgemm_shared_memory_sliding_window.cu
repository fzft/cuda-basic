/*
 * 矩阵乘法
 * 矩阵A(m, k) * 矩阵B(k, n) = 矩阵C(m, n)
 * 矩阵C的计算方式：C[i][j] = sum(A[i][k] * B[k][j]) for k in range(k)
 * 矩阵C的计算方式：C[i][j] = sum(A[i][k] * B[k][j]) for k in range(k)
 * 矩阵C的计算方式：C[i][j] = sum(A[i][k] * B[k][j]) for k in range(k)
 * 矩阵C的计算方式：C[i][j] = sum(A[i][k] * B[k][j]) for k in range(k)
 */

#include <iostream>
#include <hip/hip_runtime.h>

#include <cstdlib>

using namespace std;

void random_matrix(int rows, int cols, float *h_A) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            h_A[(i) * (cols) + (j)]  = rand() / (float)RAND_MAX;
        }
    }
} 

void cpu_sgemm(float *A, float *B, float *C, const int M, const int N, const int K) {
    for (int n = 0; n < N; n++) {
        for (int m = 0; m < M; m++) {
            for (int k = 0; k < K; k++) {
                C[m * N + n] += A[m * K + k] * B[k * N + n];
            }
        }
    }
} 

float compare_matrix(float *A, float *B, const int M, const int N) {
    int i, j;
    float max_diff = 0.0f, diff;
    int printed = 0;
    for (i = 0; i < M; i++) {
        for (j = 0; j < N; j++) {
            diff = abs(A[i * N + j] - B[i * N + j]);
            max_diff = max(max_diff, diff);
            if (printed == 0) {
                if (max_diff > 0.5f || max_diff < -0.5f){
                    cout << "A[" << i << "][" << j << "] = " << A[i * N + j] << ", B[" << i << "][" << j << "] = " << B[i * N + j] << ", diff = " << diff << endl;
                    printed++;
                }
            }
        }
    }
    return max_diff;
}

/*
C[y][x]= 
k=0
∑
K−1
​
 A[y][k]⋅B[k][x]
*/

template<unsigned int BLOCK_SIZE>
__global__ void sgemm_kernel(float *A, float *B, float *C, const int M, const int N, const int K) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    float *A_row_start = A + blockIdx.y * blockDim.y * K; 
    float *B_col_start = B + blockIdx.x * blockDim.x;

    __shared__ float shared_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float shared_B[BLOCK_SIZE][BLOCK_SIZE];

    float sum = 0.0f;
    for (int s = 0; s < K; s+= BLOCK_SIZE) {
        shared_A[threadIdx.y][threadIdx.x] = A_row_start[threadIdx.y * K + s + threadIdx.x];
        shared_B[threadIdx.y][threadIdx.x] = B_col_start[(threadIdx.y + s) * N + threadIdx.x];
        __syncthreads();
        for (int k = 0; k < BLOCK_SIZE; k++) {
            sum += shared_A[threadIdx.y][k] * shared_B[k][threadIdx.x];
        }
        __syncthreads();
    }
    C[x + y * N] = sum;
}  

int main() {
    const int m = 512;
    const int n = 512;
    const int k = 512;
    const size_t mem_size_A = m * k * sizeof(float);
    const size_t mem_size_B = k * n * sizeof(float);
    const size_t mem_size_C = m * n * sizeof(float);

    float *h_A = (float *)malloc(mem_size_A);
    float *h_B = (float *)malloc(mem_size_B);

    float *matrix_C_host_gpu_calc = (float *)malloc(mem_size_C);
    float *matrix_C_host_cpu_calc = (float *)malloc(mem_size_C);

    random_matrix(m, k, h_A);
    random_matrix(k, n, h_B);
    memset(matrix_C_host_gpu_calc, 0, mem_size_C);
    memset(matrix_C_host_cpu_calc, 0, mem_size_C); 

    float *d_A;
    float *d_B;
    float *d_C;

    hipMalloc((void**)&d_A, mem_size_A);
    hipMalloc((void**)&d_B, mem_size_B);
    hipMalloc((void**)&d_C, mem_size_C);


    hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

    cpu_sgemm(h_A, h_B, matrix_C_host_cpu_calc, m, n, k);

    const int threads_per_block = 16;
    dim3 block(threads_per_block, threads_per_block);
    dim3 grid((m + threads_per_block - 1) / threads_per_block, (n + threads_per_block - 1) / threads_per_block);
    

    sgemm_kernel<threads_per_block><<<grid, block>>>(d_A, d_B, d_C, m, n, k);
    hipMemcpy(matrix_C_host_gpu_calc, d_C, mem_size_C, hipMemcpyDeviceToHost);

    float diff = compare_matrix(matrix_C_host_gpu_calc, matrix_C_host_cpu_calc, m, n);
    if (diff > 0.5f || diff < -0.5f) {
        cout << "diff: " << diff << endl;
    } else {
        cout << "success" << endl;
    }
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    return 0;
}